/*
 *  Use cuda functions to indicate device information.
 */
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"

int main(int argc, char **argv) {
  
  // shows how many SMs on our device, among other things
  getDeviceInformation();   

  return 0;
}