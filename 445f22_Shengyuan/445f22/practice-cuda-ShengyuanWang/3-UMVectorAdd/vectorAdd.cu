#include "hip/hip_runtime.h"
//
// Demonstration using a single 1D grid and 1D block size
//
#include <math.h>   // ceil function
#include <stdio.h>  // printf

#include <hip/hip_runtime.h>
// helper functions and utilities 
#include "helper_add.h"

// Kernel function based on 1D grid of 1D blocks of threads
// In this version, thread number is:
//  its block number in the grid (blockIdx.x) times 
// the threads per block plus which thread it is in that block.
//
// This thread id is then the index into the 1D array of floats.
// This represents the simplest type of mapping:
// Each thread takes care of one element of the result
__global__ void vecAdd(float *x, float *y, int n)
{
    // Get our global thread ID
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        y[id] = x[id] + y[id];
}


int main(int argc, char **argv)
{
  printf("Vector addition using managed memory.\n");
  // Set up size of arrays for vectors
  // int N = 1<<20;
  // same value, shown as multiple of 1024, 
  // which is divisible by 32 (size of the SPs on SM)
  int N = 1024*1024;   
  printf("size (N) of 1D arrays are: %d\n\n", N);
  // host vectors
  float *x, *y;

   // Size, in bytes, of each vector; just use below
  //size_t bytes = N*sizeof(float);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  initialize(x, y, N);  // set values in each vector

  // Number of threads in each thread block
  int blockSize = 1024;
 
  // Number of thread blocks in grid needs to be based on array size
  int gridSize = (int)ceil((float)N/blockSize);
 
  printf("add vectors on device\n");
  // Execute the kernel
  vecAdd<<<gridSize, blockSize>>>(x, y, N);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  checkForErrors(y, N);

  printf("execution complete\n");

  // free unified memory
  hipFree(x);
  hipFree(y);

  return 0;

}